
#include <hip/hip_runtime.h>
__global__ void depth_conv_small(const float * __restrict__ bottom_data,float *top_data, const float *__restrict__ weights,
                                int channels,int kernel_single_size,int spatial_dim_in,int spatial_dim_out,
                                int spatial_dim_add_padding,int padding,int stride)
{   

    int kernel_size = kernel_single_size*kernel_single_size;
    extern __shared__ float bottom_data_shared[];
    __shared__ float weights_shared[9];

    const int warpid   = threadIdx.x / 32; 
    const int warp_num = blockDim.x  / 32;  
    const int laneid   = threadIdx.x % 32; 
    const int offset   = blockIdx.x * spatial_dim_in * spatial_dim_in; 

    for(int i = threadIdx.x;i<spatial_dim_add_padding*spatial_dim_add_padding;i+=128){
        bottom_data_shared[i] = 0.f;
    }
    __syncthreads();

    for( int i = warpid; i < spatial_dim_in; i += warp_num )
    {
        if( laneid < spatial_dim_in ){
            bottom_data_shared[spatial_dim_add_padding + padding + spatial_dim_add_padding*i + laneid] = __ldg(bottom_data+offset + spatial_dim_in*i + laneid);
        }
    }

    int weights_index = (blockIdx.x%channels)*kernel_size;
    if(threadIdx.x<kernel_size) weights_shared[threadIdx.x] = __ldg(weights+weights_index+threadIdx.x);

    __syncthreads();

    int top_index = blockIdx.x*spatial_dim_out*spatial_dim_out;
    float sum = 0;
    for(int i = warpid;i < spatial_dim_out;i += warp_num)
    {
        int index = laneid*stride;
        if( index <= spatial_dim_add_padding-kernel_single_size){
            sum  = bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index]        * weights_shared[0];
            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+1]      * weights_shared[1];
            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+2]      * weights_shared[2];

            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+spatial_dim_add_padding]     * weights_shared[3];
            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+spatial_dim_add_padding+1]   * weights_shared[4];
            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+spatial_dim_add_padding+2]   * weights_shared[5];

            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+spatial_dim_add_padding*2]   * weights_shared[6];
            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+spatial_dim_add_padding*2+1] * weights_shared[7];
            sum += bottom_data_shared[stride*padding*i*spatial_dim_add_padding+index+spatial_dim_add_padding*2+2] * weights_shared[8];
            
            top_data[top_index + spatial_dim_out*i + laneid] = sum;

        }
    }
}

__global__ void depth_conv_big(const float *__restrict__ bottom_data_gpu, float *top_data_gpu, const float *__restrict__ weights_gpu,
                                int channels,int kernel_single_size,
                                int spatial_dim_in,int spatial_dim_out,int spatial_dim_add_padding,
                                int padding,int stride)

{
    extern __shared__ float bottom_data_shared[]; 
    __shared__ float weights_shared[9]; 

    const int tidx = threadIdx.y * blockDim.x + threadIdx.x;
    for(int i = tidx; i<spatial_dim_add_padding*kernel_single_size; i += blockDim.x * blockDim.y) {
        bottom_data_shared[i] = 0.f;
    } 
    __syncthreads();

    int tid = (blockIdx.x/spatial_dim_out)*spatial_dim_in*spatial_dim_in;
    int height_index = (blockIdx.x % spatial_dim_out) * stride + threadIdx.y - padding; //-1

    if((unsigned int)height_index < spatial_dim_in) {
        for(int w = threadIdx.x; w < spatial_dim_in; w += blockDim.x)
        {
            bottom_data_shared[threadIdx.y * spatial_dim_add_padding + w + padding] = __ldg(bottom_data_gpu + tid + height_index * spatial_dim_in + w);
        }
    }
    
    if( threadIdx.y == 0 && threadIdx.x < 9 )
    {
        int threadblock_index_per_batch = blockIdx.x % (channels * spatial_dim_out);
        weights_shared[threadIdx.x] = __ldg(weights_gpu + ( threadblock_index_per_batch / spatial_dim_out ) * 9 + threadIdx.x);
    }
    __syncthreads();

    float sum = 0.f;
    for(int i = threadIdx.x * stride; i <= spatial_dim_add_padding-kernel_single_size; i += stride * blockDim.x) {
        sum  = bottom_data_shared[threadIdx.y * spatial_dim_add_padding + i]     * weights_shared[threadIdx.y * 3];
        sum += bottom_data_shared[threadIdx.y * spatial_dim_add_padding + i + 1] * weights_shared[threadIdx.y * 3 + 1];
        sum += bottom_data_shared[threadIdx.y * spatial_dim_add_padding + i + 2] * weights_shared[threadIdx.y * 3 + 2];


        atomicAdd(top_data_gpu + (blockIdx.x/spatial_dim_out)*spatial_dim_out*spatial_dim_out + (blockIdx.x%spatial_dim_out)*spatial_dim_out + i / stride,sum);
    }
}



/*               
     depth_conv_big<<<mobilenet_channels*out_w,dim3(32, 3),(w+2*pad)*kernelsize*sizeof(float)>>>
                (bottom_data,top_data,weights_gpu,mobilenet_channels,size,w,out_w,(w+2*pad),pad,stride);
*/

/*
    depth_conv_small<<<mobilenet_channels,128,(w+2*pad)*(w+2*pad)*sizeof(float)>>>
                (bottom_data,top_data,weights_gpu,mobilenet_channels,size,w,out_w,(w+2*pad),pad,stride);

*/
